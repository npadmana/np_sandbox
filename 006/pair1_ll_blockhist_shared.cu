#include "hip/hip_runtime.h"
// GPU paircounting 
//  -- histograms are accumulated block by block (in global memory)
//  -- buffer the second particle data into shared memory in chunks
#include <iostream>
#include <vector>
#include <cmath>
#include <ctime>
#include <numeric>
#include "book.h"

const int Nstart = 1200;
const int Ntimes = 4;
const int Nhist = 100;


#define HISTINT long long
// Set the shared memory buffer
#define BUFFER 256  

using namespace std;

// Define a particle storage class
struct Particles {
  vector<float> x, y, z;
  vector<int> w;
  int N;
};


struct ParticlesGPU {
  float *x, *y, *z;
  int *w;
  int N;
};

void AllocCopyGPU(Particles &p, ParticlesGPU &p2) {
  // Figure out how much to pad with
  p2.N = ((p.N + BUFFER - 1)/BUFFER) * BUFFER; 

  // x
  HANDLE_ERROR( hipMalloc ( (void**)&p2.x, p2.N * sizeof(float)));
  HANDLE_ERROR( hipMemcpy ( p2.x, &p.x[0], p.N * sizeof(float), hipMemcpyHostToDevice));

  // y
  HANDLE_ERROR( hipMalloc ( (void**)&p2.y, p2.N * sizeof(float)));
  HANDLE_ERROR( hipMemcpy ( p2.y, &p.y[0], p.N * sizeof(float), hipMemcpyHostToDevice));


  // z
  HANDLE_ERROR( hipMalloc ( (void**)&p2.z, p2.N * sizeof(float)));
  HANDLE_ERROR( hipMemcpy ( p2.z, &p.z[0], p.N * sizeof(float), hipMemcpyHostToDevice));

  // w
  HANDLE_ERROR( hipMalloc ( (void**)&p2.w, p2.N * sizeof(int)));
  HANDLE_ERROR( hipMemset ( p2.w, 0, p2.N * sizeof(int)));
  HANDLE_ERROR( hipMemcpy ( p2.w, &p.w[0], p.N * sizeof(int), hipMemcpyHostToDevice));
}

void FreeGPU(ParticlesGPU &p) {
  hipFree(p.x);
  hipFree(p.y);
  hipFree(p.z);
}


void makeRandomParticles(int N, Particles &p) {
  // Set number of particles
  p.N = N;

  // Resize the vectors
  p.x.resize(N);
  p.y.resize(N);
  p.z.resize(N);
  p.w.resize(N);

  // Fill in the vectors
  for (int ii=0; ii < N; ++ii) {
    p.x[ii] = float(rand())/float(RAND_MAX);
    p.y[ii] = float(rand())/float(RAND_MAX);
    p.z[ii] = float(rand())/float(RAND_MAX);
    p.w[ii] = 1;
  }

};

// Define the GPU kernel here 
__global__ void paircount_kernel(
    int N1, float *x1, float *y1, float *z1, int *w1, 
    int N2, float *x2, float *y2, float *z2, int *w2,
    int Nh, HISTINT *hist) {
  
  // Define shared memory buffer
  __shared__ float bx[BUFFER], by[BUFFER], bz[BUFFER];
  __shared__ int bw[BUFFER];


  // We distribute p1, but loop through all of p2
  int ii, jj, kk, idr;
  int stride = blockDim.x * gridDim.x;
  float x, y, z, dx, dy, dz, dr;
  int w;
  ii = threadIdx.x + blockIdx.x * blockDim.x;
  int offset = blockIdx.x * Nh;

  while (ii < N1) {
    x = x1[ii]; y = y1[ii]; z = z1[ii]; w = w1[ii];
    for (jj = 0; jj < (N2 + BUFFER - 1)/BUFFER; ++jj) {
      // Fill the buffer
      // We assume that we will pad the array out to the nearest BUFFER multiple
      // This avoids a number of internal checks
      if (threadIdx.x < BUFFER) {
        bx[threadIdx.x] = x2[jj*BUFFER + threadIdx.x];
        by[threadIdx.x] = y2[jj*BUFFER + threadIdx.x];
        bz[threadIdx.x] = z2[jj*BUFFER + threadIdx.x];
        bw[threadIdx.x] = w2[jj*BUFFER + threadIdx.x];
      }
      __syncthreads();

      for (kk=0; kk < BUFFER; ++kk) {
        dx = bx[kk] - x;
        dy = by[kk] - y;
        dz = bz[kk] - z;
        dr = sqrtf(dx*dx + dy*dy + dz*dz);
        idr = (int) (dr*Nh);
        // THE LINE BELOW IS WRONG! USE FOR SPEED TESTING ONLY!
        //if (idr < Nh) hist[idr+offset] += (w * bw[kk]);
        // THE LINE BELOW IS CORRECT
        if (idr < Nh) atomicAdd( (unsigned long long*) &hist[idr + offset], (unsigned long long) (w * bw[kk]));
      }
    }
    ii += stride;
  }

}

// Define the histogram summing kernel here 
__global__ void reduce_histogram(int Nh, HISTINT *hist) {
  int ii = threadIdx.x ;
  int offset = blockIdx.x * Nh;
  if (blockIdx.x > 0)  {
    while (ii < Nh) {
      atomicAdd( (unsigned long long*) &hist[ii], (unsigned long long) hist[ii+offset]);
      ii += blockDim.x;
    }
  }
}



// Remove all the blocking --- it's fine if this is slow-ish
void cpu_paircount(const Particles &p1, const Particles &p2, vector<HISTINT>& hist) {
  float x1, y1, z1, dx, dy, dz, dr;
  int idr, w1;
  for (int ii =0; ii < p1.N; ++ii) {
    x1 = p1.x[ii]; y1 = p1.y[ii]; z1 = p1.z[ii]; w1 = p1.w[ii]; 
    for (int jj=0; jj < p2.N; ++jj) { 
      dx = p2.x[jj]-x1;
      dy = p2.y[jj]-y1;
      dz = p2.z[jj]-z1;
      dr = sqrt(dx*dx + dy*dy + dz*dz);
      idr = (int)(dr*Nhist);
      if (idr < Nhist) hist[idr] += w1*p2.w[jj];
    }
  }
}

double cpu_harness(int N, int blocks) {
  Particles p1,p2;
  ParticlesGPU pg1, pg2;
  clock_t t0;
  double dt;
  float gpu_dt;


  cout << "Starting harness with N=" << N << endl;

  // Initialize
  t0 = clock();
  makeRandomParticles(N, p1);
  makeRandomParticles(N, p2);
  dt = difftime(clock(), t0)/double(CLOCKS_PER_SEC);
  cout << "  Time to initialize: " << dt << endl;

  // Set up GPU timers
  hipEvent_t     start, stop;
  HANDLE_ERROR( hipEventCreate( &start ) );
  HANDLE_ERROR( hipEventCreate( &stop ) );

  // Move data to GPU
  HANDLE_ERROR( hipEventRecord( start, 0 ) );
  AllocCopyGPU(p1, pg1);
  AllocCopyGPU(p2, pg2);
  HANDLE_ERROR( hipEventRecord( stop, 0 ) );
  HANDLE_ERROR( hipEventSynchronize( stop ) );
  HANDLE_ERROR( hipEventElapsedTime( &gpu_dt,
                                      start, stop ) );
  cout << "  Time to move data on to GPU (ms): " << gpu_dt << endl;


  // Set up the gpu_hist
  HISTINT *gpu_hist;

  HANDLE_ERROR( hipMalloc( (void**)&gpu_hist, Nhist*blocks*sizeof(HISTINT)));
  HANDLE_ERROR( hipMemset( gpu_hist, 0, Nhist*blocks*sizeof(HISTINT)));
  HANDLE_ERROR( hipEventRecord( start, 0 ) );
  paircount_kernel<<<blocks, 512>>>(pg1.N, pg1.x, pg1.y, pg1.z, pg1.w,
      pg2.N, pg2.x, pg2.y, pg2.z, pg2.w, Nhist, gpu_hist);
  HANDLE_ERROR( hipEventRecord( stop, 0 ) );
  HANDLE_ERROR( hipEventSynchronize( stop ) );
  HANDLE_ERROR( hipEventElapsedTime( &gpu_dt,
                                      start, stop ) );
  cout << "  Time for GPU paircounts (ms): " << gpu_dt << endl;
  // reduce histogram
  HANDLE_ERROR( hipEventRecord( start, 0 ) );
  reduce_histogram<<<blocks, 512>>>(Nhist, gpu_hist);
  HANDLE_ERROR( hipEventRecord( stop, 0 ) );
  HANDLE_ERROR( hipEventSynchronize( stop ) );
  HANDLE_ERROR( hipEventElapsedTime( &gpu_dt,
                                      start, stop ) );
  cout << "  Time to reduce GPU paircounts (ms): " << gpu_dt << endl;

  // Suck back the histogram array
  vector<HISTINT> hist1(Nhist);
  HANDLE_ERROR( hipMemcpy( &hist1[0], gpu_hist, Nhist*sizeof(HISTINT), hipMemcpyDeviceToHost));


  // Clean up
  hipFree(gpu_hist);
  FreeGPU(pg1); FreeGPU(pg2);


  // Clean up GPU timers
  HANDLE_ERROR( hipEventDestroy( start ) );
  HANDLE_ERROR( hipEventDestroy( stop ) );

  // CPU paircounting
  vector<HISTINT> hist(Nhist,0);
  t0 = clock();
  cpu_paircount(p1, p2, hist);
  dt = difftime(clock(), t0)/double(CLOCKS_PER_SEC);
  cout << "  Time to count pairs v2: " << dt << endl;

  //for (int ii = 0; ii < Nhist; ++ii) {
  //  cout << ii << " " << hist[ii] << " "  << hist1[ii] << endl;
  //}

  // Now compare histograms
  HISTINT dhist = 0, error = 0, eval = 0;
  for (int ii =0; ii < Nhist; ++ii) {
    dhist = abs(hist[ii] - hist1[ii]);
    if (dhist > error) {
      error = dhist;
      eval = hist[ii];
    }
  }
  cout << "  Difference in histograms : " << error << " " << eval << endl;

  return dt;
}


int main() {
  double timing[Ntimes];
  int i, N1;

  cout << "Pair counting timing code...." << endl;

  // kernel launch - 2x the number of mps gave best timing
  hipDeviceProp_t  prop;
  HANDLE_ERROR( hipGetDeviceProperties( &prop, 0 ) );
  int blocks = prop.multiProcessorCount * 2;
  cout << "Using blocks = " << blocks << endl;

  for (i=0, N1=Nstart; i < Ntimes; ++i, N1*=2) {
    timing[i] = cpu_harness(N1, blocks);
  }

}


#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <cmath>
#include <ctime>
#include <numeric>
#include "book.h"


// Definitions!
const int Nstart = 1000;
const int Ntimes = 4;
const int Nhist = 1000;
const int blockfac = 2;

#define HISTINT long long
#define NTHREADS 512
#define BUFHIST 512


using namespace std;

// Define a particle storage class
struct Particles {
  vector<float> x, y, z;
  int N;
};


struct ParticlesGPU {
  float *x, *y, *z;
};

void AllocCopyGPU(Particles &p, ParticlesGPU &p2) {
  // x
  HANDLE_ERROR( hipMalloc ( (void**)&p2.x, p.N * sizeof(float)));
  HANDLE_ERROR( hipMemcpy ( p2.x, &p.x[0], p.N * sizeof(float), hipMemcpyHostToDevice));

  // y
  HANDLE_ERROR( hipMalloc ( (void**)&p2.y, p.N * sizeof(float)));
  HANDLE_ERROR( hipMemcpy ( p2.y, &p.y[0], p.N * sizeof(float), hipMemcpyHostToDevice));


  // z
  HANDLE_ERROR( hipMalloc ( (void**)&p2.z, p.N * sizeof(float)));
  HANDLE_ERROR( hipMemcpy ( p2.z, &p.z[0], p.N * sizeof(float), hipMemcpyHostToDevice));
}

void FreeGPU(ParticlesGPU &p) {
  hipFree(p.x);
  hipFree(p.y);
  hipFree(p.z);
}


void makeRandomParticles(int N, Particles &p) {
  // Set number of particles
  p.N = N;

  // Resize the vectors
  p.x.resize(N);
  p.y.resize(N);
  p.z.resize(N);

  // Fill in the vectors
  for (int ii=0; ii < N; ++ii) {
    p.x[ii] = float(rand())/float(RAND_MAX);
    p.y[ii] = float(rand())/float(RAND_MAX);
    p.z[ii] = float(rand())/float(RAND_MAX);
  }

};

// Define the GPU kernel here 
__global__ void paircount_kernel(
    int N1, float *x1, float *y1, float *z1, 
    int N2, float *x2, float *y2, float *z2, 
    int Nh, HISTINT *hist) {

  // Keep a shared copy of the histogram
  __shared__ long long _hist[BUFHIST];

  // We distribute p1, but loop through all of p2
  int ii, jj, idr, nh1, ih, hstart, hend;
  int stride = blockDim.x * gridDim.x;
  float x, y, z, dx, dy, dz, dr;

  // Compute the number of histograms
  nh1 = (Nh + BUFHIST - 1)/BUFHIST;

  // Do each piece of the histogram separately
  for (ih = 0; ih < nh1; ++ih) {
    // Define histogram piece
    hstart = ih*BUFHIST;
    hend = hstart + BUFHIST;
    if (hend > Nh) hend = Nh;


    // Zero histogram
    ii = threadIdx.x;
    while (ii < BUFHIST) {
      _hist[ii] = 0ll;
      ii += blockDim.x;
    }
    __syncthreads();

    ii = threadIdx.x + blockIdx.x * blockDim.x;
    while (ii < N1) {
      x = x1[ii]; y = y1[ii]; z = z1[ii];
      for (jj = 0; jj < N2; ++jj) {
        dx = x2[jj] - x;
        dy = y2[jj] - y;
        dz = z2[jj] - z;
        dr = sqrtf(dx*dx + dy*dy + dz*dz);
        idr = (int) (dr*Nh);
        if ((idr < hend) && (idr >= hstart)) atomicAdd( (unsigned long long*) &_hist[idr-hstart], 1ll);
      }
      ii += stride;
    }

    // Synchronize
    __syncthreads();

    // Copy histogram 
    ii = threadIdx.x + hstart;
    while (ii < hend) {
      atomicAdd( (unsigned long long*) &hist[ii], _hist[ii-hstart]);
      ii += blockDim.x;
    }
    __syncthreads();
  }

}


void cpu_paircount_v2(const Particles &p1, const Particles &p2, vector<HISTINT>& hist) {
  float x1, y1, z1, dx, dy, dz;
  const int nblock=10;
  float dr[nblock];
  int idr;
  for (int ii =0; ii < p1.N; ++ii) {
    x1 = p1.x[ii]; y1 = p1.y[ii]; z1 = p1.z[ii];
    for (int jj=0; jj < p2.N/nblock; ++jj) { 
     for (int kk=0; kk < nblock; ++kk) {
      dx = p2.x[jj*nblock+kk]-x1;
      dy = p2.y[jj*nblock+kk]-y1;
      dz = p2.z[jj*nblock+kk]-z1;
      dr[kk] = sqrt(dx*dx + dy*dy + dz*dz);
     }
    

     for (int kk=0; kk < nblock; ++kk) {
       idr = (int)(dr[kk]*Nhist);
       if (idr < Nhist) hist[idr]++;
     }


    }
  }
}

double cpu_harness(int N, int blocks) {
  Particles p1,p2;
  ParticlesGPU pg1, pg2;
  clock_t t0;
  double dt;
  float gpu_dt;


  cout << "Starting harness with N=" << N << endl;

  // Initialize
  t0 = clock();
  makeRandomParticles(N, p1);
  makeRandomParticles(N, p2);
  dt = difftime(clock(), t0)/double(CLOCKS_PER_SEC);
  cout << "  Time to initialize: " << dt << endl;

  // Set up GPU timers
  hipEvent_t     start, stop;
  HANDLE_ERROR( hipEventCreate( &start ) );
  HANDLE_ERROR( hipEventCreate( &stop ) );

  // Move data to GPU
  HANDLE_ERROR( hipEventRecord( start, 0 ) );
  AllocCopyGPU(p1, pg1);
  AllocCopyGPU(p2, pg2);
  HANDLE_ERROR( hipEventRecord( stop, 0 ) );
  HANDLE_ERROR( hipEventSynchronize( stop ) );
  HANDLE_ERROR( hipEventElapsedTime( &gpu_dt,
                                      start, stop ) );
  cout << "  Time to move data on to GPU (ms): " << gpu_dt << endl;


  // Set up the gpu_hist
  HISTINT *gpu_hist;

  HANDLE_ERROR( hipMalloc( (void**)&gpu_hist, Nhist*sizeof(HISTINT)));
  HANDLE_ERROR( hipMemset( gpu_hist, 0, Nhist*sizeof(HISTINT)));
  HANDLE_ERROR( hipEventRecord( start, 0 ) );
  paircount_kernel<<<blocks, NTHREADS>>>(N, pg1.x, pg1.y, pg1.z, 
      N, pg2.x, pg2.y, pg2.z, Nhist, gpu_hist);
  HANDLE_ERROR( hipEventRecord( stop, 0 ) );
  HANDLE_ERROR( hipEventSynchronize( stop ) );
  HANDLE_ERROR( hipEventElapsedTime( &gpu_dt,
                                      start, stop ) );
  cout << "  Time for GPU paircounts (ms): " << gpu_dt << endl;
  // reduce histogram
  HANDLE_ERROR( hipEventRecord( start, 0 ) );

  // Suck back the histogram array
  vector<HISTINT> hist1(Nhist);
  HANDLE_ERROR( hipMemcpy( &hist1[0], gpu_hist, Nhist*sizeof(HISTINT), hipMemcpyDeviceToHost));


  // Clean up
  hipFree(gpu_hist);
  FreeGPU(pg1); FreeGPU(pg2);


  // Clean up GPU timers
  HANDLE_ERROR( hipEventDestroy( start ) );
  HANDLE_ERROR( hipEventDestroy( stop ) );

  // CPU paircounting
  vector<HISTINT> hist(Nhist,0);
  t0 = clock();
  cpu_paircount_v2(p1, p2, hist);
  dt = difftime(clock(), t0)/double(CLOCKS_PER_SEC);
  cout << "  Time to count pairs v2: " << dt << endl;

  //for (int ii = 0; ii < Nhist; ++ii) {
  //  cout << ii << " " << hist[ii] << " "  << hist1[ii] << endl;
  //}

  // Now compare histograms
  HISTINT dhist = 0, error = 0, eval = 0;
  for (int ii =0; ii < Nhist; ++ii) {
    dhist = abs(hist[ii] - hist1[ii]);
    if (dhist > error) {
      error = dhist;
      eval = hist[ii];
    }
  }
  cout << "  Difference in histograms : " << error << " " << eval << endl;

  return dt;
}


int main() {
  double timing[Ntimes];
  int i, N1;

  cout << "Pair counting timing code...." << endl;

  // kernel launch - 2x the number of mps gave best timing
  hipDeviceProp_t  prop;
  HANDLE_ERROR( hipGetDeviceProperties( &prop, 0 ) );
  int blocks = prop.multiProcessorCount * blockfac;
  cout << "Using blocks = " << blocks << endl;

  for (i=0, N1=Nstart; i < Ntimes; ++i, N1*=2) {
    timing[i] = cpu_harness(N1, blocks);
  }

}

